#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1048576
#define STREAM_COUNT 4

__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        C[idx] = A[idx] + B[idx];
}

int main() {
    size_t size = N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A[STREAM_COUNT], *d_B[STREAM_COUNT], *d_C[STREAM_COUNT];

    hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, size, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, size, hipHostMallocDefault);

    for (int i = 0; i < N; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    hipStream_t streams[STREAM_COUNT];
    for (int i = 0; i < STREAM_COUNT; ++i)
        hipStreamCreate(&streams[i]);

    size_t streamSize = N / STREAM_COUNT;
    size_t streamBytes = streamSize * sizeof(float);

    for (int i = 0; i < STREAM_COUNT; ++i) {
        hipMalloc((void **)&d_A[i], streamBytes);
        hipMalloc((void **)&d_B[i], streamBytes);
        hipMalloc((void **)&d_C[i], streamBytes);
    }

    for (int i = 0; i < STREAM_COUNT; ++i) {
        int offset = i * streamSize;

        hipMemcpyAsync(d_A[i], h_A + offset, streamBytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B[i], h_B + offset, streamBytes, hipMemcpyHostToDevice, streams[i]);

        int threadsPerBlock = 256;
        int blocksPerGrid = (streamSize + threadsPerBlock - 1) / threadsPerBlock;
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(d_A[i], d_B[i], d_C[i], streamSize);

        hipMemcpyAsync(h_C + offset, d_C[i], streamBytes, hipMemcpyDeviceToHost, streams[i]);
    }

    for (int i = 0; i < STREAM_COUNT; ++i)
        hipStreamSynchronize(streams[i]);

    for (int i = 0; i < STREAM_COUNT; ++i) {
        hipFree(d_A[i]); hipFree(d_B[i]); hipFree(d_C[i]);
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);

    return 0;
}
