#include "hip/hip_runtime.h"
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1048576 

__global__ void vectorScale(float *d_data, float scalar, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
        d_data[idx] *= scalar;
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);

    int worldRank, worldSize;
    MPI_Comm_rank(MPI_COMM_WORLD, &worldRank);
    MPI_Comm_size(MPI_COMM_WORLD, &worldSize);

    if (worldSize < 2) {
        printf("This example requires at least two processes.\n");
        MPI_Finalize();
        return 0;
    }

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(worldRank % deviceCount);

    size_t size = N * sizeof(float);
    float *h_data;
    float *d_data;

    hipHostAlloc((void **)&h_data, size, hipHostMallocDefault);
    hipMalloc((void **)&d_data, size);

    if (worldRank == 0) {
        for (int i = 0; i < N; ++i)
            h_data[i] = rand() / (float)RAND_MAX;
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    if (worldRank == 0) {
        hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream);

        hipStreamSynchronize(stream);

        MPI_Send(d_data, N, MPI_FLOAT, 1, 0, MPI_COMM_WORLD);
    } else if (worldRank == 1) {
        MPI_Recv(d_data, N, MPI_FLOAT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        vectorScale<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_data, 2.0f, N);

        hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream);

        hipStreamSynchronize(stream);

        printf("Data received and processed on rank 1.\n");
    }

    hipHostFree(h_data);
    hipFree(d_data);
    hipStreamDestroy(stream);

    MPI_Finalize();
    return 0;
}
